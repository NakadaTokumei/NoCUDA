
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdint>

__global__
void TestKernel(
	uint32_t* out_arr
)
{
	uint32_t i = 0;

	for(; i < threadIdx.x; i++)
		out_arr[threadIdx.x] += i;
}

int main(int argc, char** argv)
{
	uint32_t* arr = nullptr;
	uint32_t arr_host[100] = {0,};

	std::cout << "Hello, CUDA" << std::endl;

	hipMalloc(reinterpret_cast<void**>(&arr), sizeof(uint32_t)*100);

	TestKernel<<<1, 100>>>(arr);

	hipMemcpy(arr_host, arr, sizeof(uint32_t)*100, hipMemcpyDeviceToHost);

	hipFree(arr);

	for(uint32_t i = 0; i < 100; i++)
		std::cout << i << " : " << arr_host[i] << std::endl;
}
